/* Copyright (c) 2016 - 2019, the adamantine authors.
 *
 * This file is subject to the Modified BSD License and may not be distributed
 * without copyright and license information. Please refer to the file LICENSE
 * for the text and further information on this license.
 */

#define BOOST_TEST_MODULE ThermalOperator

#include <Geometry.hh>
#include <ThermalOperator.hh>

#include <deal.II/dofs/dof_tools.h>
#include <deal.II/lac/dynamic_sparsity_pattern.h>
#include <deal.II/lac/sparse_matrix.h>
#include <deal.II/numerics/matrix_tools.h>

#include <boost/property_tree/ptree.hpp>

#include "main.cc"

BOOST_AUTO_TEST_CASE(thermal_operator)
{
  MPI_Comm communicator = MPI_COMM_WORLD;

  // Create the Geometry
  boost::property_tree::ptree geometry_database;
  geometry_database.put("length", 12);
  geometry_database.put("length_divisions", 4);
  geometry_database.put("height", 6);
  geometry_database.put("height_divisions", 5);
  adamantine::Geometry<2> geometry(communicator, geometry_database);
  // Create the DoFHandler
  dealii::FE_Q<2> fe(2);
  dealii::DoFHandler<2> dof_handler(geometry.get_triangulation());
  dof_handler.distribute_dofs(fe);
  dealii::AffineConstraints<double> affine_constraints;
  affine_constraints.close();
  dealii::QGauss<1> quad(3);

  // Create the MaterialProperty
  boost::property_tree::ptree mat_prop_database;
  mat_prop_database.put("n_materials", 1);
  mat_prop_database.put("material_0.solid.density", 1.);
  mat_prop_database.put("material_0.powder.density", 1.);
  mat_prop_database.put("material_0.liquid.density", 1.);
  mat_prop_database.put("material_0.solid.specific_heat", 1.);
  mat_prop_database.put("material_0.powder.specific_heat", 1.);
  mat_prop_database.put("material_0.liquid.specific_heat", 1.);
  mat_prop_database.put("material_0.solid.thermal_conductivity", 10.);
  mat_prop_database.put("material_0.powder.thermal_conductivity", 10.);
  mat_prop_database.put("material_0.liquid.thermal_conductivity", 10.);
  std::shared_ptr<adamantine::MaterialProperty<2>> mat_properties(
      new adamantine::MaterialProperty<2>(
          communicator, geometry.get_triangulation(), mat_prop_database));

  // Initialize the ThermalOperator
  adamantine::ThermalOperator<2, 2, double> thermal_operator(communicator,
                                                             mat_properties);
  thermal_operator.setup_dofs(dof_handler, affine_constraints, quad);
  thermal_operator.reinit(dof_handler, affine_constraints);
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> dummy(thermal_operator.m());
  thermal_operator.evaluate_material_properties(dummy);
  BOOST_CHECK(thermal_operator.m() == 99);
  BOOST_CHECK(thermal_operator.m() == thermal_operator.n());

  // Check matrix-vector multiplications
  double const tolerance = 1e-15;
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> src;
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> dst_1;
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> dst_2;

  dealii::CUDAWrappers::MatrixFree<2, double> const &matrix_free =
      thermal_operator.get_matrix_free();
//TODO
/*  matrix_free.initialize_dof_vector(src);
  matrix_free.initialize_dof_vector(dst_1);
  matrix_free.initialize_dof_vector(dst_2);*/

  src = 1.;
  thermal_operator.vmult(dst_1, src);
  BOOST_CHECK_CLOSE(dst_1.l1_norm(), 0., tolerance);

  thermal_operator.Tvmult(dst_2, src);
  BOOST_CHECK_CLOSE(dst_2.l1_norm(), dst_1.l1_norm(), tolerance);

  dst_2 = 1.;
  thermal_operator.vmult_add(dst_2, src);
  thermal_operator.vmult(dst_1, src);
  dst_1 += src;
  BOOST_CHECK_CLOSE(dst_1.l1_norm(), dst_2.l1_norm(), tolerance);

  dst_1 = 1.;
  thermal_operator.Tvmult_add(dst_1, src);
  BOOST_CHECK_CLOSE(dst_1.l1_norm(), dst_2.l1_norm(), tolerance);
}

BOOST_AUTO_TEST_CASE(spmv)
{
  MPI_Comm communicator = MPI_COMM_WORLD;

  // Create the Geometry
  boost::property_tree::ptree geometry_database;
  geometry_database.put("length", 12);
  geometry_database.put("length_divisions", 4);
  geometry_database.put("height", 6);
  geometry_database.put("height_divisions", 5);
  adamantine::Geometry<2> geometry(communicator, geometry_database);
  // Create the DoFHandler
  dealii::FE_Q<2> fe(2);
  dealii::DoFHandler<2> dof_handler(geometry.get_triangulation());
  dof_handler.distribute_dofs(fe);
  dealii::AffineConstraints<double> affine_constraints;
  affine_constraints.close();
  dealii::QGauss<1> quad(3);

  // Create the MaterialProperty
  boost::property_tree::ptree mat_prop_database;
  mat_prop_database.put("n_materials", 1);
  mat_prop_database.put("material_0.solid.density", 1.);
  mat_prop_database.put("material_0.powder.density", 1.);
  mat_prop_database.put("material_0.liquid.density", 1.);
  mat_prop_database.put("material_0.solid.specific_heat", 1.);
  mat_prop_database.put("material_0.powder.specific_heat", 1.);
  mat_prop_database.put("material_0.liquid.specific_heat", 1.);
  mat_prop_database.put("material_0.solid.thermal_conductivity", 1.);
  mat_prop_database.put("material_0.powder.thermal_conductivity", 1.);
  mat_prop_database.put("material_0.liquid.thermal_conductivity", 1.);
  std::shared_ptr<adamantine::MaterialProperty<2>> mat_properties(
      new adamantine::MaterialProperty<2>(
          communicator, geometry.get_triangulation(), mat_prop_database));

  // Initialize the ThermalOperator
  adamantine::ThermalOperator<2, 2, double> thermal_operator(communicator,
                                                             mat_properties);
  thermal_operator.setup_dofs(dof_handler, affine_constraints, quad);
  thermal_operator.reinit(dof_handler, affine_constraints);
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> dummy(thermal_operator.m());
  thermal_operator.evaluate_material_properties(dummy);
  BOOST_CHECK(thermal_operator.m() == 99);
  BOOST_CHECK(thermal_operator.m() == thermal_operator.n());

  // Build the matrix. This only works in serial.
  dealii::DynamicSparsityPattern dsp(dof_handler.n_dofs());
  dealii::DoFTools::make_sparsity_pattern(dof_handler, dsp, affine_constraints);
  dealii::SparsityPattern sparsity_pattern;
  sparsity_pattern.copy_from(dsp);
  dealii::SparseMatrix<double> sparse_matrix(sparsity_pattern);
  dealii::MatrixCreator::create_laplace_matrix(
      dof_handler, dealii::QGauss<2>(3), sparse_matrix);

  // Compare vmult using matrix free and building the matrix
  double const tolerance = 1e-12;
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> src;
  dealii::LA::distributed::Vector<double> src_host;
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> dst_1;
  dealii::LA::distributed::Vector<double> dst_1_host;
  dealii::LA::distributed::Vector<double, dealii::MemorySpace::CUDA> dst_2;
  dealii::LA::distributed::Vector<double> dst_2_host;

  dealii::CUDAWrappers::MatrixFree<2, double> const &matrix_free =
      thermal_operator.get_matrix_free();
  matrix_free.initialize_dof_vector(src);
  matrix_free.initialize_dof_vector(dst_1);
  matrix_free.initialize_dof_vector(dst_2);
  src_host.reinit(src.get_partitioner());
  dst_2_host.reinit(dst_2.get_partitioner());
  dst_1_host.reinit(dst_1.get_partitioner());

  for (unsigned int i = 0; i < thermal_operator.m(); ++i)
  {
    src_host = 0.;
    src_host[i] = 1;
    src.import(src_host, dealii::VectorOperation::insert);
    thermal_operator.vmult(dst_1, src);
    dst_1_host.import(dst_1, dealii::VectorOperation::insert);
    sparse_matrix.vmult(dst_2_host, src_host);
    for (unsigned int j = 0; j < thermal_operator.m(); ++j)
      BOOST_CHECK_CLOSE(dst_1_host[j], -dst_2_host[j], tolerance);
  }
}
